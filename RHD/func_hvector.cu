
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>
double funct_H(double n, double p, double u, double v, double w, double k, int i)
{
   double a;
   if(i == 0)
   {
      a = n*w/sqrt(-pow(w,2.0)-pow(v,2.0)-pow(u,2.0)+1);
   }
   else if(i == 1)
   {
      a = -(k*p+(k-1)*n)*w/((k-1)*pow(w,2.0)+(k-1)*pow(v,2.0)+(k-1)*pow(u,2.0)-k+1);
   }
   else if(i == 2)
   {
      a = -(k*p+(k-1)*n)*u*w/((k-1)*pow(w,2.0)+(k-1)*pow(v,2.0)+(k-1)*pow(u,2.0)-k+1);
   }
   else if(i == 3)
   {
      a = -(k*p+(k-1)*n)*v*w/((k-1)*pow(w,2.0)+(k-1)*pow(v,2.0)+(k-1)*pow(u,2.0)-k+1);
   }
   else if(i == 4)
   {
      a = -((p+(k-1)*n)*pow(w,2.0)+(1-k)*p*pow(v,2.0)+(1-k)*p*pow(u,2.0)+(k-1)*p)/((k-1)*pow(w,2.0)+(k-1)*pow(v,2.0)+(k-1)*pow(u,2.0)-k+1);
   }
   return a;
}
