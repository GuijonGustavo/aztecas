
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>
double funct_F(double n, double p, double u, double v, double w, double k, int i)
{
   double a;
   if(i == 0)
   {
      a = n*u/sqrt(-pow(w,2.0)-pow(v,2.0)-pow(u,2.0)+1);
   }
   else if(i == 1)
   {
      a = -(k*p+(k-1)*n)*u/((k-1)*pow(w,2.0)+(k-1)*pow(v,2.0)+(k-1)*pow(u,2.0)-k+1);
   }
   else if(i == 2)
   {
      a = ((k-1)*p*pow(w,2.0)+(k-1)*p*pow(v,2.0)+((1-k)*n-p)*pow(u,2.0)+(1-k)*p)/((k-1)*pow(w,2.0)+(k-1)*pow(v,2.0)+(k-1)*pow(u,2.0)-k+1);
   }
   else if(i == 3)
   {
      a = -(k*p+(k-1)*n)*u*v/((k-1)*pow(w,2.0)+(k-1)*pow(v,2.0)+(k-1)*pow(u,2.0)-k+1);
   }
   else if(i == 4)
   {
      a = -(k*p+(k-1)*n)*u*w/((k-1)*pow(w,2.0)+(k-1)*pow(v,2.0)+(k-1)*pow(u,2.0)-k+1);
   }
   return a;
}
