
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>
double funct_Q(double n, double p, double u, double v, double w, double k, int i)
{
   double a;
   if(i == 0)
   {
      a = n/sqrt(-pow(w,2.0)-pow(v,2.0)-pow(u,2.0)+1);
   }
   else if(i == 1)
   {
      a = -((k-1)*p*pow(w,2.0)+(k-1)*p*pow(v,2.0)+(k-1)*p*pow(u,2.0)+p+(k-1)*n)/((k-1)*pow(w,2.0)+(k-1)*pow(v,2.0)+(k-1)*pow(u,2.0)-k+1);
   }
   else if(i == 2)
   {
      a = -(k*p+(k-1)*n)*u/((k-1)*pow(w,2.0)+(k-1)*pow(v,2.0)+(k-1)*pow(u,2.0)-k+1);
   }
   else if(i == 3)
   {
      a = -(k*p+(k-1)*n)*v/((k-1)*pow(w,2.0)+(k-1)*pow(v,2.0)+(k-1)*pow(u,2.0)-k+1);
   }
   else if(i == 4)
   {
      a = -(k*p+(k-1)*n)*w/((k-1)*pow(w,2.0)+(k-1)*pow(v,2.0)+(k-1)*pow(u,2.0)-k+1);
   }
   return a;
}
