
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>
double funct_G(double n, double p, double u, double v, double w, double k, int i)
{
   double a;
   if(i == 0)
   {
      a = n*v/sqrt(-pow(w,2.0)-pow(v,2.0)-pow(u,2.0)+1);
   }
   else if(i == 1)
   {
      a = -(k*p+(k-1)*n)*v/((k-1)*pow(w,2.0)+(k-1)*pow(v,2.0)+(k-1)*pow(u,2.0)-k+1);
   }
   else if(i == 2)
   {
      a = -(k*p+(k-1)*n)*u*v/((k-1)*pow(w,2.0)+(k-1)*pow(v,2.0)+(k-1)*pow(u,2.0)-k+1);
   }
   else if(i == 3)
   {
      a = ((k-1)*p*pow(w,2.0)+((1-k)*n-p)*pow(v,2.0)+(k-1)*p*pow(u,2.0)+(1-k)*p)/((k-1)*pow(w,2.0)+(k-1)*pow(v,2.0)+(k-1)*pow(u,2.0)-k+1);
   }
   else if(i == 4)
   {
      a = -(k*p+(k-1)*n)*v*w/((k-1)*pow(w,2.0)+(k-1)*pow(v,2.0)+(k-1)*pow(u,2.0)-k+1);
   }
   return a;
}
