
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>
double funct_A(double n, double p, double u, double v, double w, double k, int i, int j)
{
   double a;
   if(i == 0 && j == 0)
   {
      a = -(((pow(k,2.0)-k)*p+(pow(k,2.0)-2*k+1)*n)*pow(w,4.0)+(((2*pow(k,2.0)-2*k)*p+(2*pow(k,2.0)-4*k+2)*n)*pow(v,2.0)+((2*pow(k,2.0)-2*k)*p+(2*pow(k,2.0)-4*k+2)*n)*pow(u,2.0)-pow(k,2.0)*p+(k-pow(k,2.0))*n)*pow(w,2.0)+((pow(k,2.0)-k)*p+(pow(k,2.0)-2*k+1)*n)*pow(v,4.0)+(((2*pow(k,2.0)-2*k)*p+(2*pow(k,2.0)-4*k+2)*n)*pow(u,2.0)-pow(k,2.0)*p+(k-pow(k,2.0))*n)*pow(v,2.0)+((pow(k,2.0)-k)*p+(pow(k,2.0)-2*k+1)*n)*pow(u,4.0)+((k-pow(k,2.0))*n-pow(k,2.0)*p)*pow(u,2.0)+k*p+(k-1)*n)/(sqrt(-pow(w,2.0)-pow(v,2.0)-pow(u,2.0)+1)*((pow(k,2.0)-k)*p*pow(w,2.0)+(pow(k,2.0)-k)*p*pow(v,2.0)+(pow(k,2.0)-k)*p*pow(u,2.0)-k*p+(1-k)*n));
   }
   else if(i == 0 && j == 1)
   {
      a = -((pow(k,2.0)-k)*n*pow(w,2.0)+(pow(k,2.0)-k)*n*pow(v,2.0)+(pow(k,2.0)-k)*n*pow(u,2.0))/((pow(k,2.0)-k)*p*pow(w,2.0)+(pow(k,2.0)-k)*p*pow(v,2.0)+(pow(k,2.0)-k)*p*pow(u,2.0)-k*p+(1-k)*n);
   }
   else if(i == 0 && j == 2)
   {
      a = ((pow(k,2.0)-2*k+1)*n*u*pow(w,2.0)+(pow(k,2.0)-2*k+1)*n*u*pow(v,2.0)+(pow(k,2.0)-2*k+1)*n*pow(u,3.0)+(k-1)*n*u)/((pow(k,2.0)-k)*p*pow(w,2.0)+(pow(k,2.0)-k)*p*pow(v,2.0)+(pow(k,2.0)-k)*p*pow(u,2.0)-k*p+(1-k)*n);
   }
   else if(i == 0 && j == 3)
   {
      a = ((pow(k,2.0)-2*k+1)*n*v*pow(w,2.0)+(pow(k,2.0)-2*k+1)*n*pow(v,3.0)+((pow(k,2.0)-2*k+1)*n*pow(u,2.0)+(k-1)*n)*v)/((pow(k,2.0)-k)*p*pow(w,2.0)+(pow(k,2.0)-k)*p*pow(v,2.0)+(pow(k,2.0)-k)*p*pow(u,2.0)-k*p+(1-k)*n);
   }
   else if(i == 0 && j == 4)
   {
      a = ((pow(k,2.0)-2*k+1)*n*pow(w,3.0)+((pow(k,2.0)-2*k+1)*n*pow(v,2.0)+(pow(k,2.0)-2*k+1)*n*pow(u,2.0)+(k-1)*n)*w)/((pow(k,2.0)-k)*p*pow(w,2.0)+(pow(k,2.0)-k)*p*pow(v,2.0)+(pow(k,2.0)-k)*p*pow(u,2.0)-k*p+(1-k)*n);
   }
   else if(i == 1 && j == 0)
   {
      a = -(((pow(k,2.0)-k)*p+(pow(k,2.0)-2*k+1)*n)*pow(w,2.0)+((pow(k,2.0)-k)*p+(pow(k,2.0)-2*k+1)*n)*pow(v,2.0)+((pow(k,2.0)-k)*p+(pow(k,2.0)-2*k+1)*n)*pow(u,2.0)+(k-pow(k,2.0))*p+(-pow(k,2.0)+2*k-1)*n)/(sqrt(-pow(w,2.0)-pow(v,2.0)-pow(u,2.0)+1)*((pow(k,2.0)-k)*p*pow(w,2.0)+(pow(k,2.0)-k)*p*pow(v,2.0)+(pow(k,2.0)-k)*p*pow(u,2.0)-k*p+(1-k)*n));
   }
   else if(i == 1 && j == 1)
   {
      a = -((pow(k,2.0)-k)*p*pow(w,2.0)+(pow(k,2.0)-k)*p*pow(v,2.0)+(pow(k,2.0)-k)*p*pow(u,2.0)+(pow(k,2.0)-k)*p+(pow(k,2.0)-2*k+1)*n)/((pow(k,2.0)-k)*p*pow(w,2.0)+(pow(k,2.0)-k)*p*pow(v,2.0)+(pow(k,2.0)-k)*p*pow(u,2.0)-k*p+(1-k)*n);
   }
   else if(i == 1 && j == 2)
   {
      a = ((2*pow(k,2.0)-2*k)*p+(pow(k,2.0)-2*k+1)*n)*u/((pow(k,2.0)-k)*p*pow(w,2.0)+(pow(k,2.0)-k)*p*pow(v,2.0)+(pow(k,2.0)-k)*p*pow(u,2.0)-k*p+(1-k)*n);
   }
   else if(i == 1 && j == 3)
   {
      a = ((2*pow(k,2.0)-2*k)*p+(pow(k,2.0)-2*k+1)*n)*v/((pow(k,2.0)-k)*p*pow(w,2.0)+(pow(k,2.0)-k)*p*pow(v,2.0)+(pow(k,2.0)-k)*p*pow(u,2.0)-k*p+(1-k)*n);
   }
   else if(i == 1 && j == 4)
   {
      a = ((2*pow(k,2.0)-2*k)*p+(pow(k,2.0)-2*k+1)*n)*w/((pow(k,2.0)-k)*p*pow(w,2.0)+(pow(k,2.0)-k)*p*pow(v,2.0)+(pow(k,2.0)-k)*p*pow(u,2.0)-k*p+(1-k)*n);
   }
   else if(i == 2 && j == 0)
   {
      a = -((pow(k,2.0)-2*k+1)*u*pow(w,4.0)+((2*pow(k,2.0)-4*k+2)*u*pow(v,2.0)+(2*pow(k,2.0)-4*k+2)*pow(u,3.0)+(-2*pow(k,2.0)+4*k-2)*u)*pow(w,2.0)+(pow(k,2.0)-2*k+1)*u*pow(v,4.0)+((2*pow(k,2.0)-4*k+2)*pow(u,3.0)+(-2*pow(k,2.0)+4*k-2)*u)*pow(v,2.0)+(pow(k,2.0)-2*k+1)*pow(u,5.0)+(-2*pow(k,2.0)+4*k-2)*pow(u,3.0)+(pow(k,2.0)-2*k+1)*u)/(sqrt(-pow(w,2.0)-pow(v,2.0)-pow(u,2.0)+1)*((pow(k,2.0)-k)*p*pow(w,2.0)+(pow(k,2.0)-k)*p*pow(v,2.0)+(pow(k,2.0)-k)*p*pow(u,2.0)-k*p+(1-k)*n));
   }
   else if(i == 2 && j == 1)
   {
      a = -((pow(k,2.0)-k)*u*pow(w,2.0)+(pow(k,2.0)-k)*u*pow(v,2.0)+(pow(k,2.0)-k)*pow(u,3.0)+(k-pow(k,2.0))*u)/((pow(k,2.0)-k)*p*pow(w,2.0)+(pow(k,2.0)-k)*p*pow(v,2.0)+(pow(k,2.0)-k)*p*pow(u,2.0)-k*p+(1-k)*n);
   }
   else if(i == 2 && j == 2)
   {
      a = -((pow(k,3.0)-2*pow(k,2.0)+k)*p*pow(w,4.0)+((2*pow(k,3.0)-4*pow(k,2.0)+2*k)*p*pow(v,2.0)+(-pow(k,3.0)+3*pow(k,2.0)-3*k+1)*n*pow(u,2.0)+(pow(k,2.0)-pow(k,3.0))*p+(-pow(k,2.0)+2*k-1)*n)*pow(w,2.0)+(pow(k,3.0)-2*pow(k,2.0)+k)*p*pow(v,4.0)+((-pow(k,3.0)+3*pow(k,2.0)-3*k+1)*n*pow(u,2.0)+(pow(k,2.0)-pow(k,3.0))*p+(-pow(k,2.0)+2*k-1)*n)*pow(v,2.0)+((-pow(k,3.0)+2*pow(k,2.0)-k)*p+(-pow(k,3.0)+3*pow(k,2.0)-3*k+1)*n)*pow(u,4.0)+((pow(k,3.0)-3*pow(k,2.0)+2*k)*p+(pow(k,3.0)-4*pow(k,2.0)+5*k-2)*n)*pow(u,2.0)+(pow(k,2.0)-k)*p+(pow(k,2.0)-2*k+1)*n)/(((pow(k,3.0)-pow(k,2.0))*pow(p,2.0)+(pow(k,3.0)-2*pow(k,2.0)+k)*n*p)*pow(w,2.0)+((pow(k,3.0)-pow(k,2.0))*pow(p,2.0)+(pow(k,3.0)-2*pow(k,2.0)+k)*n*p)*pow(v,2.0)+((pow(k,3.0)-pow(k,2.0))*pow(p,2.0)+(pow(k,3.0)-2*pow(k,2.0)+k)*n*p)*pow(u,2.0)-pow(k,2.0)*pow(p,2.0)+(2*k-2*pow(k,2.0))*n*p+(-pow(k,2.0)+2*k-1)*pow(n,2.0));
   }
   else if(i == 2 && j == 3)
   {
      a = (((2*pow(k,3.0)-4*pow(k,2.0)+2*k)*p+(pow(k,3.0)-3*pow(k,2.0)+3*k-1)*n)*u*v*pow(w,2.0)+((2*pow(k,3.0)-4*pow(k,2.0)+2*k)*p+(pow(k,3.0)-3*pow(k,2.0)+3*k-1)*n)*u*pow(v,3.0)+(((2*pow(k,3.0)-4*pow(k,2.0)+2*k)*p+(pow(k,3.0)-3*pow(k,2.0)+3*k-1)*n)*pow(u,3.0)+((-2*pow(k,3.0)+4*pow(k,2.0)-2*k)*p+(-pow(k,3.0)+3*pow(k,2.0)-3*k+1)*n)*u)*v)/(((pow(k,3.0)-pow(k,2.0))*pow(p,2.0)+(pow(k,3.0)-2*pow(k,2.0)+k)*n*p)*pow(w,2.0)+((pow(k,3.0)-pow(k,2.0))*pow(p,2.0)+(pow(k,3.0)-2*pow(k,2.0)+k)*n*p)*pow(v,2.0)+((pow(k,3.0)-pow(k,2.0))*pow(p,2.0)+(pow(k,3.0)-2*pow(k,2.0)+k)*n*p)*pow(u,2.0)-pow(k,2.0)*pow(p,2.0)+(2*k-2*pow(k,2.0))*n*p+(-pow(k,2.0)+2*k-1)*pow(n,2.0));
   }
   else if(i == 2 && j == 4)
   {
      a = (((2*pow(k,3.0)-4*pow(k,2.0)+2*k)*p+(pow(k,3.0)-3*pow(k,2.0)+3*k-1)*n)*u*pow(w,3.0)+(((2*pow(k,3.0)-4*pow(k,2.0)+2*k)*p+(pow(k,3.0)-3*pow(k,2.0)+3*k-1)*n)*u*pow(v,2.0)+((2*pow(k,3.0)-4*pow(k,2.0)+2*k)*p+(pow(k,3.0)-3*pow(k,2.0)+3*k-1)*n)*pow(u,3.0)+((-2*pow(k,3.0)+4*pow(k,2.0)-2*k)*p+(-pow(k,3.0)+3*pow(k,2.0)-3*k+1)*n)*u)*w)/(((pow(k,3.0)-pow(k,2.0))*pow(p,2.0)+(pow(k,3.0)-2*pow(k,2.0)+k)*n*p)*pow(w,2.0)+((pow(k,3.0)-pow(k,2.0))*pow(p,2.0)+(pow(k,3.0)-2*pow(k,2.0)+k)*n*p)*pow(v,2.0)+((pow(k,3.0)-pow(k,2.0))*pow(p,2.0)+(pow(k,3.0)-2*pow(k,2.0)+k)*n*p)*pow(u,2.0)-pow(k,2.0)*pow(p,2.0)+(2*k-2*pow(k,2.0))*n*p+(-pow(k,2.0)+2*k-1)*pow(n,2.0));
   }
   else if(i == 3 && j == 0)
   {
      a = -((pow(k,2.0)-2*k+1)*v*pow(w,4.0)+((2*pow(k,2.0)-4*k+2)*pow(v,3.0)+((2*pow(k,2.0)-4*k+2)*pow(u,2.0)-2*pow(k,2.0)+4*k-2)*v)*pow(w,2.0)+(pow(k,2.0)-2*k+1)*pow(v,5.0)+((2*pow(k,2.0)-4*k+2)*pow(u,2.0)-2*pow(k,2.0)+4*k-2)*pow(v,3.0)+((pow(k,2.0)-2*k+1)*pow(u,4.0)+(-2*pow(k,2.0)+4*k-2)*pow(u,2.0)+pow(k,2.0)-2*k+1)*v)/(sqrt(-pow(w,2.0)-pow(v,2.0)-pow(u,2.0)+1)*((pow(k,2.0)-k)*p*pow(w,2.0)+(pow(k,2.0)-k)*p*pow(v,2.0)+(pow(k,2.0)-k)*p*pow(u,2.0)-k*p+(1-k)*n));
   }
   else if(i == 3 && j == 1)
   {
      a = -((pow(k,2.0)-k)*v*pow(w,2.0)+(pow(k,2.0)-k)*pow(v,3.0)+((pow(k,2.0)-k)*pow(u,2.0)-pow(k,2.0)+k)*v)/((pow(k,2.0)-k)*p*pow(w,2.0)+(pow(k,2.0)-k)*p*pow(v,2.0)+(pow(k,2.0)-k)*p*pow(u,2.0)-k*p+(1-k)*n);
   }
   else if(i == 3 && j == 2)
   {
      a = (((2*pow(k,3.0)-4*pow(k,2.0)+2*k)*p+(pow(k,3.0)-3*pow(k,2.0)+3*k-1)*n)*u*v*pow(w,2.0)+((2*pow(k,3.0)-4*pow(k,2.0)+2*k)*p+(pow(k,3.0)-3*pow(k,2.0)+3*k-1)*n)*u*pow(v,3.0)+(((2*pow(k,3.0)-4*pow(k,2.0)+2*k)*p+(pow(k,3.0)-3*pow(k,2.0)+3*k-1)*n)*pow(u,3.0)+((-2*pow(k,3.0)+4*pow(k,2.0)-2*k)*p+(-pow(k,3.0)+3*pow(k,2.0)-3*k+1)*n)*u)*v)/(((pow(k,3.0)-pow(k,2.0))*pow(p,2.0)+(pow(k,3.0)-2*pow(k,2.0)+k)*n*p)*pow(w,2.0)+((pow(k,3.0)-pow(k,2.0))*pow(p,2.0)+(pow(k,3.0)-2*pow(k,2.0)+k)*n*p)*pow(v,2.0)+((pow(k,3.0)-pow(k,2.0))*pow(p,2.0)+(pow(k,3.0)-2*pow(k,2.0)+k)*n*p)*pow(u,2.0)-pow(k,2.0)*pow(p,2.0)+(2*k-2*pow(k,2.0))*n*p+(-pow(k,2.0)+2*k-1)*pow(n,2.0));
   }
   else if(i == 3 && j == 3)
   {
      a = -((pow(k,3.0)-2*pow(k,2.0)+k)*p*pow(w,4.0)+((-pow(k,3.0)+3*pow(k,2.0)-3*k+1)*n*pow(v,2.0)+(2*pow(k,3.0)-4*pow(k,2.0)+2*k)*p*pow(u,2.0)+(pow(k,2.0)-pow(k,3.0))*p+(-pow(k,2.0)+2*k-1)*n)*pow(w,2.0)+((-pow(k,3.0)+2*pow(k,2.0)-k)*p+(-pow(k,3.0)+3*pow(k,2.0)-3*k+1)*n)*pow(v,4.0)+((-pow(k,3.0)+3*pow(k,2.0)-3*k+1)*n*pow(u,2.0)+(pow(k,3.0)-3*pow(k,2.0)+2*k)*p+(pow(k,3.0)-4*pow(k,2.0)+5*k-2)*n)*pow(v,2.0)+(pow(k,3.0)-2*pow(k,2.0)+k)*p*pow(u,4.0)+((pow(k,2.0)-pow(k,3.0))*p+(-pow(k,2.0)+2*k-1)*n)*pow(u,2.0)+(pow(k,2.0)-k)*p+(pow(k,2.0)-2*k+1)*n)/(((pow(k,3.0)-pow(k,2.0))*pow(p,2.0)+(pow(k,3.0)-2*pow(k,2.0)+k)*n*p)*pow(w,2.0)+((pow(k,3.0)-pow(k,2.0))*pow(p,2.0)+(pow(k,3.0)-2*pow(k,2.0)+k)*n*p)*pow(v,2.0)+((pow(k,3.0)-pow(k,2.0))*pow(p,2.0)+(pow(k,3.0)-2*pow(k,2.0)+k)*n*p)*pow(u,2.0)-pow(k,2.0)*pow(p,2.0)+(2*k-2*pow(k,2.0))*n*p+(-pow(k,2.0)+2*k-1)*pow(n,2.0));
   }
   else if(i == 3 && j == 4)
   {
      a = (((2*pow(k,3.0)-4*pow(k,2.0)+2*k)*p+(pow(k,3.0)-3*pow(k,2.0)+3*k-1)*n)*v*pow(w,3.0)+(((2*pow(k,3.0)-4*pow(k,2.0)+2*k)*p+(pow(k,3.0)-3*pow(k,2.0)+3*k-1)*n)*pow(v,3.0)+(((2*pow(k,3.0)-4*pow(k,2.0)+2*k)*p+(pow(k,3.0)-3*pow(k,2.0)+3*k-1)*n)*pow(u,2.0)+(-2*pow(k,3.0)+4*pow(k,2.0)-2*k)*p+(-pow(k,3.0)+3*pow(k,2.0)-3*k+1)*n)*v)*w)/(((pow(k,3.0)-pow(k,2.0))*pow(p,2.0)+(pow(k,3.0)-2*pow(k,2.0)+k)*n*p)*pow(w,2.0)+((pow(k,3.0)-pow(k,2.0))*pow(p,2.0)+(pow(k,3.0)-2*pow(k,2.0)+k)*n*p)*pow(v,2.0)+((pow(k,3.0)-pow(k,2.0))*pow(p,2.0)+(pow(k,3.0)-2*pow(k,2.0)+k)*n*p)*pow(u,2.0)-pow(k,2.0)*pow(p,2.0)+(2*k-2*pow(k,2.0))*n*p+(-pow(k,2.0)+2*k-1)*pow(n,2.0));
   }
   else if(i == 4 && j == 0)
   {
      a = -((pow(k,2.0)-2*k+1)*pow(w,5.0)+((2*pow(k,2.0)-4*k+2)*pow(v,2.0)+(2*pow(k,2.0)-4*k+2)*pow(u,2.0)-2*pow(k,2.0)+4*k-2)*pow(w,3.0)+((pow(k,2.0)-2*k+1)*pow(v,4.0)+((2*pow(k,2.0)-4*k+2)*pow(u,2.0)-2*pow(k,2.0)+4*k-2)*pow(v,2.0)+(pow(k,2.0)-2*k+1)*pow(u,4.0)+(-2*pow(k,2.0)+4*k-2)*pow(u,2.0)+pow(k,2.0)-2*k+1)*w)/(sqrt(-pow(w,2.0)-pow(v,2.0)-pow(u,2.0)+1)*((pow(k,2.0)-k)*p*pow(w,2.0)+(pow(k,2.0)-k)*p*pow(v,2.0)+(pow(k,2.0)-k)*p*pow(u,2.0)-k*p+(1-k)*n));
   }
   else if(i == 4 && j == 1)
   {
      a = -((pow(k,2.0)-k)*pow(w,3.0)+((pow(k,2.0)-k)*pow(v,2.0)+(pow(k,2.0)-k)*pow(u,2.0)-pow(k,2.0)+k)*w)/((pow(k,2.0)-k)*p*pow(w,2.0)+(pow(k,2.0)-k)*p*pow(v,2.0)+(pow(k,2.0)-k)*p*pow(u,2.0)-k*p+(1-k)*n);
   }
   else if(i == 4 && j == 2)
   {
      a = (((2*pow(k,3.0)-4*pow(k,2.0)+2*k)*p+(pow(k,3.0)-3*pow(k,2.0)+3*k-1)*n)*u*pow(w,3.0)+(((2*pow(k,3.0)-4*pow(k,2.0)+2*k)*p+(pow(k,3.0)-3*pow(k,2.0)+3*k-1)*n)*u*pow(v,2.0)+((2*pow(k,3.0)-4*pow(k,2.0)+2*k)*p+(pow(k,3.0)-3*pow(k,2.0)+3*k-1)*n)*pow(u,3.0)+((-2*pow(k,3.0)+4*pow(k,2.0)-2*k)*p+(-pow(k,3.0)+3*pow(k,2.0)-3*k+1)*n)*u)*w)/(((pow(k,3.0)-pow(k,2.0))*pow(p,2.0)+(pow(k,3.0)-2*pow(k,2.0)+k)*n*p)*pow(w,2.0)+((pow(k,3.0)-pow(k,2.0))*pow(p,2.0)+(pow(k,3.0)-2*pow(k,2.0)+k)*n*p)*pow(v,2.0)+((pow(k,3.0)-pow(k,2.0))*pow(p,2.0)+(pow(k,3.0)-2*pow(k,2.0)+k)*n*p)*pow(u,2.0)-pow(k,2.0)*pow(p,2.0)+(2*k-2*pow(k,2.0))*n*p+(-pow(k,2.0)+2*k-1)*pow(n,2.0));
   }
   else if(i == 4 && j == 3)
   {
      a = (((2*pow(k,3.0)-4*pow(k,2.0)+2*k)*p+(pow(k,3.0)-3*pow(k,2.0)+3*k-1)*n)*v*pow(w,3.0)+(((2*pow(k,3.0)-4*pow(k,2.0)+2*k)*p+(pow(k,3.0)-3*pow(k,2.0)+3*k-1)*n)*pow(v,3.0)+(((2*pow(k,3.0)-4*pow(k,2.0)+2*k)*p+(pow(k,3.0)-3*pow(k,2.0)+3*k-1)*n)*pow(u,2.0)+(-2*pow(k,3.0)+4*pow(k,2.0)-2*k)*p+(-pow(k,3.0)+3*pow(k,2.0)-3*k+1)*n)*v)*w)/(((pow(k,3.0)-pow(k,2.0))*pow(p,2.0)+(pow(k,3.0)-2*pow(k,2.0)+k)*n*p)*pow(w,2.0)+((pow(k,3.0)-pow(k,2.0))*pow(p,2.0)+(pow(k,3.0)-2*pow(k,2.0)+k)*n*p)*pow(v,2.0)+((pow(k,3.0)-pow(k,2.0))*pow(p,2.0)+(pow(k,3.0)-2*pow(k,2.0)+k)*n*p)*pow(u,2.0)-pow(k,2.0)*pow(p,2.0)+(2*k-2*pow(k,2.0))*n*p+(-pow(k,2.0)+2*k-1)*pow(n,2.0));
   }
   else if(i == 4 && j == 4)
   {
      a = (((pow(k,3.0)-2*pow(k,2.0)+k)*p+(pow(k,3.0)-3*pow(k,2.0)+3*k-1)*n)*pow(w,4.0)+((pow(k,3.0)-3*pow(k,2.0)+3*k-1)*n*pow(v,2.0)+(pow(k,3.0)-3*pow(k,2.0)+3*k-1)*n*pow(u,2.0)+(-pow(k,3.0)+3*pow(k,2.0)-2*k)*p+(-pow(k,3.0)+4*pow(k,2.0)-5*k+2)*n)*pow(w,2.0)+(-pow(k,3.0)+2*pow(k,2.0)-k)*p*pow(v,4.0)+((-2*pow(k,3.0)+4*pow(k,2.0)-2*k)*p*pow(u,2.0)+(pow(k,3.0)-pow(k,2.0))*p+(pow(k,2.0)-2*k+1)*n)*pow(v,2.0)+(-pow(k,3.0)+2*pow(k,2.0)-k)*p*pow(u,4.0)+((pow(k,3.0)-pow(k,2.0))*p+(pow(k,2.0)-2*k+1)*n)*pow(u,2.0)+(k-pow(k,2.0))*p+(-pow(k,2.0)+2*k-1)*n)/(((pow(k,3.0)-pow(k,2.0))*pow(p,2.0)+(pow(k,3.0)-2*pow(k,2.0)+k)*n*p)*pow(w,2.0)+((pow(k,3.0)-pow(k,2.0))*pow(p,2.0)+(pow(k,3.0)-2*pow(k,2.0)+k)*n*p)*pow(v,2.0)+((pow(k,3.0)-pow(k,2.0))*pow(p,2.0)+(pow(k,3.0)-2*pow(k,2.0)+k)*n*p)*pow(u,2.0)-pow(k,2.0)*pow(p,2.0)+(2*k-2*pow(k,2.0))*n*p+(-pow(k,2.0)+2*k-1)*pow(n,2.0));
   }
   return a;
}
