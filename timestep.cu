#include "hip/hip_runtime.h"
/*
 *   This program is free software: you can redistribute it and/or modify
 *   it under the terms of the GNU General Public License as published by
 *   the Free Software Foundation, either version 3 of the License, or
 *   (at your option) any later version.
 *
 *   This program is distributed in the hope that it will be useful,
 *   but WITHOUT ANY WARRANTY; without even the implied warranty of
 *   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *   GNU General Public License for more details.
 *
 *   You should have received a copy of the GNU General Public License
 *   along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

//Do not erase any of these libraries//
#include<stdio.h>
#include<math.h>
#include<stdlib.h>
#include<string.h>
#include"./Headers/main.h"

#define min(a,b) (((a)<(b))?(a):(b))
#define max(a,b) (((a)>(b))?(a):(b))

extern par_ par;

double TIMESTEP()
{
	//int i;
	//int eq  = par.eq;
	//int Nx  = par.xmesh;
	//int dim = par.dim;
	//double k = par.k;
	double cou = par.cou;
	//double dt, vmax, dtmin, c;
	
	return cou*0.0001;	
}
