
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>
double funct_Mn(double n, double p, double u, double v, double w, double k, int i)
{
   double a;
   if(i == 0)
   {
      a = pow(v,2.0)+pow(n,2.0);
   }
   else if(i == 1)   {
      a = pow(v,2.0)+pow(k,2.0)*pow(p,2.0);
   }
   else if(i == 2)   {
      a = pow(v,2.0);
   }
   else if(i == 3)   {
      a = (pow(n,2.0)*pow(v,2.0)+1)/pow(n,2.0);
   }
   else if(i == 4)   {
      a = pow(v,2.0);
   }
   return a;
}
