
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>
double funct_F(double n, double p, double u, double v, double w, double k, int i)
{
   double a;
   if(i == 0)
   {
      a = n*u;
   }
   else if(i == 1)
   {
      a = ((k-1)*n*u*pow(w,2.0)+(k-1)*n*u*pow(v,2.0)+(k-1)*n*pow(u,3.0)+2*k*p*u)/(2*k-2);
   }
   else if(i == 2)
   {
      a = n*pow(u,2.0)+p;
   }
   else if(i == 3)
   {
      a = n*u*v;
   }
   else if(i == 4)
   {
      a = n*u*w;
   }
   return a;
}
