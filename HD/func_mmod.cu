
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>
double funct_Mm(double n, double p, double u, double v, double w, double k, int i)
{
   double a;
   if(i == 0)
   {
      a = pow(u,2.0)+pow(n,2.0);
   }
   else if(i == 1)   {
      a = pow(u,2.0)+pow(k,2.0)*pow(p,2.0);
   }
   else if(i == 2)   {
      a = (pow(n,2.0)*pow(u,2.0)+1)/pow(n,2.0);
   }
   else if(i == 3)   {
      a = pow(u,2.0);
   }
   else if(i == 4)   {
      a = pow(u,2.0);
   }
   return a;
}
