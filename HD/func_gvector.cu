
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>
double funct_G(double n, double p, double u, double v, double w, double k, int i)
{
   double a;
   if(i == 0)
   {
      a = n*v;
   }
   else if(i == 1)
   {
      a = ((k-1)*n*v*pow(w,2.0)+(k-1)*n*pow(v,3.0)+((k-1)*n*pow(u,2.0)+2*k*p)*v)/(2*k-2);
   }
   else if(i == 2)
   {
      a = n*u*v;
   }
   else if(i == 3)
   {
      a = n*pow(v,2.0)+p;
   }
   else if(i == 4)
   {
      a = n*v*w;
   }
   return a;
}
