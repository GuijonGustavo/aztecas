
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>
double funct_Q(double n, double p, double u, double v, double w, double k, int i)
{
   double a;
   if(i == 0)
   {
      a = n;
   }
   else if(i == 1)
   {
      a = ((k-1)*n*pow(w,2.0)+(k-1)*n*pow(v,2.0)+(k-1)*n*pow(u,2.0)+2*p)/(2*k-2);
   }
   else if(i == 2)
   {
      a = n*u;
   }
   else if(i == 3)
   {
      a = n*v;
   }
   else if(i == 4)
   {
      a = n*w;
   }
   return a;
}
