
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>
double funct_H(double n, double p, double u, double v, double w, double k, int i)
{
   double a;
   if(i == 0)
   {
      a = n*w;
   }
   else if(i == 1)
   {
      a = ((k-1)*n*pow(w,3.0)+((k-1)*n*pow(v,2.0)+(k-1)*n*pow(u,2.0)+2*k*p)*w)/(2*k-2);
   }
   else if(i == 2)
   {
      a = n*u*w;
   }
   else if(i == 3)
   {
      a = n*v*w;
   }
   else if(i == 4)
   {
      a = n*pow(w,2.0)+p;
   }
   return a;
}
