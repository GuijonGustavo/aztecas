
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>
double funct_Mo(double n, double p, double u, double v, double w, double k, int i)
{
   double a;
   if(i == 0)
   {
      a = pow(w,2.0)+pow(n,2.0);
   }
   else if(i == 1)   {
      a = pow(w,2.0)+pow(k,2.0)*pow(p,2.0);
   }
   else if(i == 2)   {
      a = pow(w,2.0);
   }
   else if(i == 3)   {
      a = pow(w,2.0);
   }
   else if(i == 4)   {
      a = (pow(n,2.0)*pow(w,2.0)+1)/pow(n,2.0);
   }
   return a;
}
