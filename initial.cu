#include "hip/hip_runtime.h"
/*
 *   This program is free software: you can redistribute it and/or modify
 *   it under the terms of the GNU General Public License as published by
 *   the Free Software Foundation, either version 3 of the License, or
 *   (at your option) any later version.
 *
 *   This program is distributed in the hope that it will be useful,
 *   but WITHOUT ANY WARRANTY; without even the implied warranty of
 *   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *   GNU General Public License for more details.
 *
 *   You should have received a copy of the GNU General Public License
 *   along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

//Do not erase any of these libraries//
#include<stdio.h>
#include<math.h>
#include<string.h>
#include"./Headers/main.h"

extern par_ par;
extern double *X;
extern double *U;
extern double *Y;

void INITIAL(double *dx, double *dy, double *dz, double *dtprint, double *tmax)
{
	//int n, i, j, k;
	int i, j, k;
	//int eq = par.eq;
	int Nx = par.xmesh;
	int Ny = par.ymesh;
	int Nz = par.zmesh;
	int dim = par.dim;
	double xmax = par.xmax;
	//double ymax = par.ymax;
	//double zmax = par.zmax;
	double xmin = par.xmin;
	//double ymin = par.ymin;
	//double zmin = par.zmin;

	printf("\nRiemann problem\n");
	if(dim == 1)
	{
		for(i = 0; i <= Nx; i++)
		{	
			if(X[i] < (xmax - xmin)/2.0)
			{
				U[c1(0,i)] = par.n1;
				U[c1(1,i)] = par.p1;
				U[c1(2,i)] = par.vx1;
			}
			else
			{
				U[c1(0,i)] = par.n2;
				U[c1(1,i)] = par.p2;
				U[c1(2,i)] = par.vx2;
			}
		}
	}
	else if(dim == 2)
	{
/*
		FILE *fdata;
		fdata = fopen("DATOS1154","r");

		for(i = 0; i <= Nx; i++)
		{
			for(j = 0; j <= Ny; j++)
			{	
				fscanf(fdata,"%lf %lf %lf %lf %lf %lf",&X[i],&Y[j],&U[0*(Nx+1)*(Ny+1) + i*(Ny+1) + j],&U[1*(Nx+1)*(Ny+1) + i*(Ny+1) + j],&U[2*(Nx+1)*(Ny+1)+ i*(Ny+1) + j],&U[3*(Nx+1)*(Ny+1) + i*(Ny+1) + j]);
			}
		}

		fclose(fdata);
*/

///////////////////////////
//--------Jet------------//
///////////////////////////
/*	
		for(i = 0; i <= Nx; i++)
		{	
			for(j = 0; j <= Ny; j++)
			{
				if(Y[j] == 0)
				{
					if(X[i] < 5.75 && X[i] > 4.25)
					{
						U[c2(0,i,j)] = par.n1;
						U[c2(1,i,j)] = par.p1;
						U[c2(2,i,j)] = par.vx1;
						U[c2(3,i,j)] = par.vy1;
					}
					else
					{
						U[c2(0,i,j)] = par.n2;
						U[c2(1,i,j)] = par.p2;
						U[c2(2,i,j)] = par.vx2;
						U[c2(3,i,j)] = par.vy2;
					}
				}				
				else
				{
					U[c2(0,i,j)] = par.n2;
					U[c2(1,i,j)] = par.p2;
					U[c2(2,i,j)] = par.vx2;
					U[c2(3,i,j)] = par.vy2;
				}
			}
		}
*/
/////////////////////////////

///////////////////////////
//--------Sedov----------//
///////////////////////////

		for(i = 0; i <= Nx; i++)
		{	
			for(j = 0; j <= Ny; j++)
			{
				if(pow(X[i] - 0.5,2.0) + pow(Y[j] - 0.1,2.0) < pow(0.05,2.0))
				{
					U[c2(0,i,j)] = 1.0;
					U[c2(1,i,j)] = 1.0;
					U[c2(2,i,j)] = 0.0;
					U[c2(3,i,j)] = 0.0;
				}
				else
				{
					U[c2(0,i,j)] = 1.0;
					U[c2(1,i,j)] = 0.01;
					U[c2(2,i,j)] = 0.0;
					U[c2(3,i,j)] = 0.0;
				}
			}				
		}

/////////////////////////////

///////////////////////////
//-------Riemann---------//
///////////////////////////
/*
		for(i = 0; i <= Nx; i++)
		{	
			for(j = 0; j <= Ny; j++)
			{
				if(Y[j] < 0.5)
				{
					U[c2(0,i,j)] = par.n1;
					U[c2(1,i,j)] = par.p1;
					U[c2(2,i,j)] = par.vx1;
					U[c2(3,i,j)] = par.vy1;
				}
				else
				{
					U[c2(0,i,j)] = par.n2;
					U[c2(1,i,j)] = par.p2;
					U[c2(2,i,j)] = par.vx2;
					U[c2(3,i,j)] = par.vy2;
				}
			}				
		}
*/
/////////////////////////////
	}
	else if(dim == 3)
	{
		for(i = 0; i <= Nx; i++)
		{	
			for(j = 0; j <= Ny; j++)
			{
				for(k = 0; k <= Nz; k++)
				{
					if(X[i] < (xmax - xmin)/2.0)
					{
						U[c3(0,i,j,k)] = par.n1;
						U[c3(1,i,j,k)] = par.p1;
						U[c3(2,i,j,k)] = par.vx1;
						U[c3(3,i,j,k)] = par.vy1;
						U[c3(4,i,j,k)] = par.vz1;
					}
					else
					{
						U[c3(0,i,j,k)] = par.n2;
						U[c3(1,i,j,k)] = par.p2;
						U[c3(2,i,j,k)] = par.vx2;
						U[c3(3,i,j,k)] = par.vy2;
						U[c3(4,i,j,k)] = par.vz2;
					}
				}
			}
		}
	}	
	else
	{
		printf("ERROR: Variable dim distinta de {1,2,3}");
	}
	
	*tmax    = par.tmax;
	*dtprint = par.timefile;
}

