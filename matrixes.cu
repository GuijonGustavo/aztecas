#include "hip/hip_runtime.h"
/*
 *   This program is free software: you can redistribute it and/or modify
 *   it under the terms of the GNU General Public License as published by
 *   the Free Software Foundation, either version 3 of the License, or
 *   (at your option) any later version.
 *
 *   This program is distributed in the hope that it will be useful,
 *   but WITHOUT ANY WARRANTY; without even the implied warranty of
 *   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *   GNU General Public License for more details.
 *
 *   You should have received a copy of the GNU General Public License
 *   along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

//Do not erase any of these libraries//
#include<stdio.h>
#include<math.h>
#include<stdlib.h>
#include<string.h>
#include<gsl/gsl_matrix.h>
#include<gsl/gsl_blas.h>
#include"./Headers/main.h"
#include"./Headers/limiters.h"
#include"./Headers/matrix.h"
#include"./Headers/vector.h"
#include"./Headers/mod.h"

#define min(a,b) (((a)<(b))?(a):(b))
#define max(a,b) (((a)>(b))?(a):(b))

extern par_ par;
extern double *fm;
extern double *fp;
extern double *qp;
extern double *qm;

int AMATRIX1D(double *u, double *A, int i)
{
	int m, n;
	int eq  = par.eq;
	//int dim = par.dim;
	double K = par.k;
	double N, P, vx=0, vy=0, vz=0;

	N  = u[c1(0,i)];
	P  = u[c1(1,i)];
	vx = u[c1(2,i)];

	for(m = 0; m < eq; m++)
	{
		for(n = 0; n < eq; n++)
		{
			A[m*eq + n] = funct_A(N,P,vx,vy,vz,K,m,n);
		}
	}	

	return 0;
}

int AMATRIX2D(double *u, double *A, int i, int j)
{
	int m, n;
	int eq  = par.eq;
	//int dim = par.dim;
	double K = par.k;
	double N, P, vx=0, vy=0, vz=0;

	N  = u[c2(0,i,j)];
	P  = u[c2(1,i,j)];
	vx = u[c2(2,i,j)];
	vy = u[c2(3,i,j)];

	for(m = 0; m < eq; m++)
	{
		for(n = 0; n < eq; n++)
		{
			A[m*eq + n] = funct_A(N,P,vx,vy,vz,K,m,n);
		}
	}	

	return 0;
}

int AMATRIX3D(double *u, double *A, int i, int j, int k)
{
	int m, n;
	int eq  = par.eq;
	//int dim = par.dim;
	double K = par.k;
	double N, P, vx=0, vy=0, vz=0;

	N  = u[c3(0,i,j,k)];
	P  = u[c3(1,i,j,k)];
	vx = u[c3(2,i,j,k)];
	vy = u[c3(3,i,j,k)];
	vz = u[c3(4,i,j,k)];

	for(m = 0; m < eq; m++)
	{
		for(n = 0; n < eq; n++)
		{
			A[m*eq + n] = funct_A(N,P,vx,vy,vz,K,m,n);
		}
	}	

	return 0;
}

///////////////////////////////////////////////////////////////////////////

int VECTOR(double *lp, double *lm, int pm, char f)
{
	int n;
	int eq = par.eq;
	int dim = par.dim;
	double K = par.k;
	double *u, lr, ll;
   double Np, Pp, vxp=0, vyp=0, vzp=0;
   double Nm, Pm, vxm=0, vym=0, vzm=0;
	gsl_matrix *dp = gsl_matrix_alloc(3,3);
	gsl_matrix *dm = gsl_matrix_alloc(3,3);

   if(pm == 1)
   {
		switch(f)
		{
			case 'f':
				u = l->uxp;
			break;
			
			case 'g':
				u = l->uyp;
			break;
		}
   }
   else
   {
		switch(f)
		{
			case 'f':
				u = l->uxm;
			break;
			
			case 'g':
				u = l->uym;
			break;
		}
   }

	Np  = u[1*eq + 0];
	Pp  = u[1*eq + 1];
	vxp = u[1*eq + 2];

	Nm  = u[0*eq + 0];
	Pm  = u[0*eq + 1];
	vxm = u[0*eq + 2];

	if(dim == 2)
	{
		vyp = u[1*eq + 3];
		vym = u[0*eq + 3];
	}
	else if(dim == 3)
	{
		vyp = u[1*eq + 3];
		vzp = u[1*eq + 4];
		vym = u[0*eq + 3];
		vzm = u[0*eq + 4];
	}

	for(n = 0; n < eq; n++)
	{
		qp[n] = funct_Q(Np,Pp,vxp,vyp,vzp,K,n);
		qm[n] = funct_Q(Nm,Pm,vxm,vym,vzm,K,n);

		switch(f)
		{
			case 'f':
				fp[n] = funct_F(Np,Pp,vxp,vyp,vzp,K,n);
				fm[n] = funct_F(Nm,Pm,vxm,vym,vzm,K,n);
			break;

			case 'g':
				fp[n] = funct_G(Np,Pp,vxp,vyp,vzp,K,n);
				fm[n] = funct_G(Nm,Pm,vxm,vym,vzm,K,n);
			break;
		}
	}	

	for(n = 0; n < 3; n++)
	{
		switch(f)
		{
			case 'f':	
				gsl_matrix_set(dp,n,n,funct_Dm(Np,Pp,vxp,vyp,vzp,K,n,n));
				gsl_matrix_set(dm,n,n,funct_Dm(Nm,Pm,vxm,vym,vzm,K,n,n));
			break;

			case 'g':
				gsl_matrix_set(dp,n,n,funct_Dn(Np,Pp,vxp,vyp,vzp,K,n,n));
				gsl_matrix_set(dm,n,n,funct_Dn(Nm,Pm,vxm,vym,vzm,K,n,n));
			break;	
		}
	}			

	lr = gsl_matrix_max(dp);
	ll = gsl_matrix_max(dm);

	*lp = max(lr,ll);

	lr = gsl_matrix_min(dp);
	ll = gsl_matrix_min(dm);

	*lm = min(lr,ll);

	gsl_matrix_free(dp);
	gsl_matrix_free(dm);

	return 0;
}

///////////////////////////////////////////////////////////////////////////
