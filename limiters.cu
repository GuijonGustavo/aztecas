#include "hip/hip_runtime.h"
/*
 *   This program is free software: you can redistribute it and/or modify
 *   it under the terms of the GNU General Public License as published by
 *   the Free Software Foundation, either version 3 of the License, or
 *   (at your option) any later version.
 *
 *   This program is distributed in the hope that it will be useful,
 *   but WITHOUT ANY WARRANTY; without even the implied warranty of
 *   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *   GNU General Public License for more details.
 *
 *   You should have received a copy of the GNU General Public License
 *   along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

//Do not erase any of these libraries//
#include<stdio.h>
#include<math.h>
#include<stdlib.h>
#include<string.h>
#include<gsl/gsl_matrix.h>
#include<gsl/gsl_blas.h>
#include"./Headers/main.h"
#include"./Headers/limiters.h"

extern par_ par;
lim_ *l;
int RECONST1D(double *u, int i)
{
	int n;
	int eq = par.eq;
	double dup2, dup1, dum1, dum2;

	for(n = 0; n < eq; n++)
	{
		dum2 = u[c1(n,i-1)] - u[c1(n,i-2)];
		dum1 = u[c1(n,i)]   - u[c1(n,i-1)];
		dup1 = u[c1(n,i+1)] - u[c1(n,i)];
		dup2 = u[c1(n,i+2)] - u[c1(n,i+1)];

		l->sx[0*eq + n] = LIMITER(dum1,dum2);
		l->sx[1*eq + n] = LIMITER(dup1,dum1);
		l->sx[2*eq + n] = LIMITER(dup2,dup1);

		l->uxm[0*eq + n] = u[c1(n,i-1)] + 0.5*l->sx[0*eq + n];
		l->uxm[1*eq + n] = u[c1(n,i)] - 0.5*l->sx[1*eq + n];

		l->uxp[0*eq + n] = u[c1(n,i)] + 0.5*l->sx[1*eq + n];
		l->uxp[1*eq + n] = u[c1(n,i+1)] - 0.5*l->sx[2*eq + n];
	}

	return 0;
}

int RECONST2D(double *u, int i, int j)
{
	int n;
	int eq = par.eq;
	double dup2, dup1, dum1, dum2;

	for(n = 0; n < eq; n++)
	{
		dum2 = u[c2(n,i-1,j)] - u[c2(n,i-2,j)];
		dum1 = u[c2(n,i,j)]   - u[c2(n,i-1,j)];
		dup1 = u[c2(n,i+1,j)] - u[c2(n,i,j)];
		dup2 = u[c2(n,i+2,j)] - u[c2(n,i+1,j)];

		l->sx[0*eq + n] = LIMITER(dum1,dum2);
		l->sx[1*eq + n] = LIMITER(dup1,dum1);
		l->sx[2*eq + n] = LIMITER(dup2,dup1);

		l->uxm[0*eq + n] = u[c2(n,i-1,j)] + 0.5*l->sx[0*eq + n];
		l->uxm[1*eq + n] = u[c2(n,i,j)] - 0.5*l->sx[1*eq + n];

		l->uxp[0*eq + n] = u[c2(n,i,j)] + 0.5*l->sx[1*eq + n];
		l->uxp[1*eq + n] = u[c2(n,i+1,j)] - 0.5*l->sx[2*eq + n];

		dum2 = u[c2(n,i,j-1)] - u[c2(n,i,j-2)];
		dum1 = u[c2(n,i,j)]   - u[c2(n,i,j-1)];
		dup1 = u[c2(n,i,j+1)] - u[c2(n,i,j)];
		dup2 = u[c2(n,i,j+2)] - u[c2(n,i,j+1)];

		l->sy[0*eq + n] = LIMITER(dum1,dum2);
		l->sy[1*eq + n] = LIMITER(dup1,dum1);
		l->sy[2*eq + n] = LIMITER(dup2,dup1);

		l->uym[0*eq + n] = u[c2(n,i,j-1)] + 0.5*l->sy[0*eq + n];
		l->uym[1*eq + n] = u[c2(n,i,j)] - 0.5*l->sy[1*eq + n];

		l->uyp[0*eq + n] = u[c2(n,i,j)] + 0.5*l->sy[1*eq + n];
		l->uyp[1*eq + n] = u[c2(n,i,j+1)] - 0.5*l->sy[2*eq + n];
	}

	return 0;
}

double LIMITER(double A, double B)
{
	double sig;
	char lim = par.lim;

	switch(lim)
	{
		case 'G':
			sig = GODUNOV(A,B);
		break;

		case 'M':
			sig = MINMOD(A,B);
		break;

		case 'C':
			sig = MC(A,B);
		break;

		case 'S':
			sig = SUPERBEE(A,B);
		break;
	}

	return sig;
}

double MAXMOD(double A, double B)
{  
   double sig;
   
   if(A*B <= 0)
   {  
      sig = 0;
   }
   
   if(A*B > 0)
   {  
      if(fabs(A) > fabs(B))
      {
         sig = A;
      }
      
      if(fabs(A) < fabs(B))
      {
         sig = B;
      }
   }
   
   return sig;
}

double MINMOD(double A, double B)
{
   double sig;
   
   if(A*B < 0)
   {  
      sig = 0;
   }
   
   if(A*B >= 0)
   {  
      if(fabs(A) < fabs(B))
      {  
         sig = A;
      }
      
      if(fabs(A) > fabs(B))
      {  
         sig = B;
      }
   }
   
   return sig;
}

double MC(double A, double B)
{
   double sig;
   double C;

   C = (A + B)/2;

   if(A*B <= 0)
   {
      sig = 0;
   }

   if(A*B > 0)
   {
      if(fabs(A) < fabs(B) && 2*fabs(A) < fabs(C))
      {
         sig = 2*A;
      }

      if(fabs(A) > fabs(B) && 2*fabs(B) < fabs(C))
      {
         sig = 2*B;
      }

      if(fabs(C) < 2*fabs(A) && fabs(C) < 2*fabs(B))
      {
         sig= C;
      }
   }

   return sig;
}

double SUPERBEE(double A, double B)
{
   double sig1;
   double sig2;
   double sig;

   sig1 = MINMOD(2*A,B);
   sig2 = MINMOD(A,2*B);

   sig = MAXMOD(sig1,sig2);

   return sig;
}

double GODUNOV(double A, double B)
{
   double sig;

   sig = 0;

   return sig;
}
