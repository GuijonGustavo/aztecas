/*
 *   This program is free software: you can redistribute it and/or modify
 *   it under the terms of the GNU General Public License as published by
 *   the Free Software Foundation, either version 3 of the License, or
 *   (at your option) any later version.
 *
 *   This program is distributed in the hope that it will be useful,
 *   but WITHOUT ANY WARRANTY; without even the implied warranty of
 *   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *   GNU General Public License for more details.
 *
 *   You should have received a copy of the GNU General Public License
 *   along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

//Do not erase any of these libraries//
#include<stdio.h>
#include<math.h>
#include<stdlib.h>
#include<string.h>
#include <hip/hip_runtime.h>
#include "./Headers/Error.h"
#include <time.h>
#include "./Headers/GpuTimer.h"
#include "./Headers/CpuTimer.h"
#include <gsl/gsl_integration.h>
#include <omp.h>
#include"./Headers/main.h"

int main(void)
{
	ReadData(); //Read the input data write on INPUT file
	int itprint;
	double dt, dx, dy, dz;
	double time, tmax, dtprint, tprint;

	//With these functions we allocate the dynamical memory for
	//U[eq][Nx][Ny][Nz], X[Nx], Y[Ny] and Z[Nz] func_planarPOINTER.c
	allocateArray();

	//We set the mesh func_planarMESH.c
	MESH(&dx,&dy,&dz);

	//We set the initial parameters func_planarINITIAL.c
	INITIAL(&dx,&dy,&dz,&dtprint,&tmax);
	tprint  = 0.0; //Initialize printing parameter
	itprint = 0;   //Initialize file numeration
	time    = 0.0; //Initialize time
	dt      = 0.0; //Initialize dt
	
	//while(time <= tmax)
	for(;;)
	{	
		//In this part we compute the time step
		dt = TIMESTEP();
	
		//We print the values: file (DATOS*) and to terminal func_planarOUTPUT.c
		PrintValues(&time,&dx,&dy,&dz,&dt,&tprint,&dtprint,&itprint);
		
		//In here we set the integration method (Finite volume method)
		INTEGRATION(&dx,&dy,&dz,&dt,&time);
	if(time > tmax) break;
	}

//	free(U);
//	free(X);
//	free(Y);
//	free(Z);

	return 0;
}
